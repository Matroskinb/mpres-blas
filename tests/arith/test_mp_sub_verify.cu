#include "hip/hip_runtime.h"
/*
 *  Test for validating the mp_sub routines
 *
 *  Copyright 2021 by Konstantin Isupov.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */


#include "../../src/arith/sub.cuh"
#include "../../src/arith/assign.cuh"

static __global__ void testCudaSub(mp_float_ptr dz, mp_float_ptr dx, mp_float_ptr dy){
    cuda::mp_sub(dz, dx[0], dy[0]);
}

int main() {
    rns_const_init();
    mp_const_init();
    rns_const_print(true);
    rns_eval_const_print();

    mp_float_t x, y, z;
    mp_set_d(&x, 1.1e-10);
    mp_set_d(&y, 1.1e10);

    printf("\nARG X = %.16f", mp_get_d(x));
    //mp_print(&x);
    //printf("\n");

    printf("\nARG Y = %.16f", mp_get_d(y));
    //mp_print(&y);
    //printf("\n");

    //CPU SUB
    mp_sub(&z, x, y);

    //mp_print(&z);
    //printf("\n");

    mp_float_ptr dx;
    mp_float_ptr dy;
    mp_float_ptr dz;
    hipMalloc(&dx, sizeof(mp_float_t));
    hipMalloc(&dy, sizeof(mp_float_t));
    hipMalloc(&dz, sizeof(mp_float_t));
    hipMemcpy(dx, &x, sizeof(mp_float_t), hipMemcpyHostToDevice);
    hipMemcpy(dy, &y, sizeof(mp_float_t), hipMemcpyHostToDevice);

    //CUDA SUB
    testCudaSub<<<1,1>>>(dz, dx, dy);
    mp_set_d(&z, 0.0);
    hipMemcpy(&z, dz, sizeof(mp_float_t), hipMemcpyDeviceToHost);
    printf("\nCUDA RESULT = %.16f", mp_get_d(z));
    //mp_print(&z);
    //printf("\n");

    return 0;
}