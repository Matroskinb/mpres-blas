#include "hip/hip_runtime.h"
/*
 *  Performance test for BLAS GEMV routines
 *
 *  Copyright 2020 by Konstantin Isupov.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "omp.h"
#include "../../logger.cuh"
#include "../../timers.cuh"
#include "../../tsthelper.cuh"
#include "../../../src/sparse/mpspmvell.cuh"
#include "../../blas/performance/3rdparty.cuh"


#define M 300  // Number of matrix rows and the vector Y dimension
#define N 300 // Number of matrix columns and the vector X dimension
#define LDA (M) // Specifies the leading dimension of A as declared in the calling (sub)program.
#define TRANS "N" // Specifies the operation: if trans = 'N' or 'n', then y := alpha*A*x + beta*y; if trans = 'T' or 't' or 'C' or 'c' then y = alpha*A**T*x + beta*y (transposed matrix).
#define INCX 1 // Specifies the increment for the elements of x.
#define INCY 1 // Specifies the increment for the elements of y.
#define REPEAT_TEST 1 //Number of repeats

//Execution configuration for mpgemv
#define MPRES_CUDA_BLOCKS_FIELDS_ROUND 256
#define MPRES_CUDA_THREADS_FIELDS_ROUND 128
#define MPRES_CUDA_BLOCKS_RESIDUES 256
#define MPRES_CUDA_THREADS_REDUCE 32

#define OPENBLAS_THREADS 4

int MP_PRECISION_DEC; //in decimal digits
int INP_BITS; //in bits
int INP_DIGITS; //in decimal digits

void setPrecisions() {
    MP_PRECISION_DEC = (int) (MP_PRECISION / 3.32 + 1);
    INP_BITS = (int) (MP_PRECISION / 4);
    INP_DIGITS = (int) (INP_BITS / 3.32 + 1);
}

void initialize() {
    hipDeviceReset();
    rns_const_init();
    mp_const_init();
    setPrecisions();
    mp_real::mp_init(MP_PRECISION_DEC);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
}

void finalize() {
    mp_real::mp_finalize();
}

void convert_vector(mp_float_ptr dest, mpfr_t *source, int width) {
    for (int i = 0; i < width; i++) {
        mp_set_mpfr(&dest[i], source[i]);
    }
}

void convert_matrix(mp_float_ptr dest, mpfr_t *source, int rows, int cols) {
    int width = rows * cols;
    for (int i = 0; i < width; i++) {
        mp_set_mpfr(&dest[i], source[i]);
    }
}



/********************* GEMV implementations and benchmarks *********************/

/////////
// MPRES-BLAS (structure of arrays)
/////////
void mpres_test(enum mblas_trans_type trans, int m, int n, int maxNonZeros, int lenx, int leny, mp_float_t *A, int *indices,
                mp_float_t *x, mp_float_t *y) {
    InitCudaTimer();
    Logger::printDash();
    PrintTimerName("[GPU] MPRES-BLAS gemv");

    // Host data
    mp_float_ptr hx = x;
    mp_float_ptr hy = y;
    mp_float_ptr hA = A;

    //GPU data
    mp_array_t dx;
    mp_array_t dy;
    mp_array_t dA;
    mp_array_t dbuf1;
    int *dindices;

    //Init data
    cuda::mp_array_init(dx, lenx);
    cuda::mp_array_init(dy, leny);
    cuda::mp_array_init(dA, m * maxNonZeros);
    cuda::mp_array_init(dbuf1, m * n);

    hipMalloc(&dindices, sizeof(int) * m * maxNonZeros);

    //Copying to the GPU
    cuda::mp_array_host2device(dx, hx, lenx);
    cuda::mp_array_host2device(dA, hA, m * maxNonZeros);
    hipMemcpy(dindices, indices, sizeof(int) * m * maxNonZeros, hipMemcpyHostToDevice);

    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
    //Launch
    for (int i = 0; i < REPEAT_TEST; i++) {
        cuda::mp_array_host2device(dy, hy, leny);StartCudaTimer();

        cuda::spmv<
                MPRES_CUDA_BLOCKS_FIELDS_ROUND,
                MPRES_CUDA_THREADS_FIELDS_ROUND,
                MPRES_CUDA_BLOCKS_RESIDUES,
                MPRES_CUDA_THREADS_REDUCE>
                (trans, m, n, maxNonZeros, dA, dindices, dx, dy, dbuf1);
        EndCudaTimer();
    }
    PrintCudaTimer("took");
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Copying to the host
    cuda::mp_array_device2host(hy, dy, leny);
    print_mp_sum(hy, leny);

    //Cleanup
    delete[] hx;
    delete[] hy;
    delete[] hA;
    cuda::mp_array_clear(dx);
    cuda::mp_array_clear(dy);
    cuda::mp_array_clear(dA);
    cuda::mp_array_clear(dbuf1);
    hipFree(dindices);
}

void create_ellpack_matrices(char filename[], mp_float_t *&data, int *&indices, int &m, int &n, int &maxNonZeros) {

    std::ifstream file(filename);
    int num_lines = 0;

// Ignore comments headers
    while (file.peek() == '%') file.ignore(2048, '\n');

// Read number of rows and columns
    file >> m >> n >> num_lines;

// Create 2D array and fill with zeros
    int *nonZeros;
    nonZeros = new int[m]();

// fill the matrix with data
    for (int l = 0; l < num_lines; l++) {
        double fileData = 0.0;
        int row = 0, col = 0;
        file >> row >> col >> fileData;
        nonZeros[(row - 1)] = nonZeros[(row - 1)] + 1;
    }


    for (int i = 0; i < m; i++) {
        std::cout << nonZeros[i] << " ";
    }
    std::cout << std::endl;

    maxNonZeros = *std::max_element(nonZeros, nonZeros + m);

    data = new mp_float_t[m * (maxNonZeros)];
    indices = new int[m * (maxNonZeros)]();

    //курсор в начало
    file.seekg(0, ios::beg);

    // Ignore comments headers
    while (file.peek() == '%') file.ignore(2048, '\n');

    // Read number of rows and columns
    file >> m >> n >> num_lines;

    int * colNum = new int[m]();

    //разобраться как заново считывать файл
    for (int l = 0; l < num_lines; l++) {
        double fileData = 0.0;
        int row = 0, col = 0;
        file >> row >> col >> fileData;
        mp_set_d(&data[colNum[(row - 1)] * m + (row - 1)], fileData);
        indices[colNum[(row - 1)] * m + (row - 1)] = (col-1);
        colNum[row - 1]++;
    }

    file.close();

    std::cout << "data" << std::endl;
    for (int j = 0; j < m; ++j) {
        for (int i = 0; i < maxNonZeros; ++i) {
            std::cout << mp_get_d(&data[j + m * i]) << " ";
        }
        std::cout << std::endl;
    }

/*
    std::cout << "data inline" << std::endl;
    for (int i = 0; i < m * maxNonZeros; ++i) {
        std::cout << i << " = " << mp_get_d(&data[i]) << std::endl;
    }
*/

    std::cout << std::endl;
    std::cout << "indices" << std::endl;
    for (int j = 0; j < m; ++j) {
        for (int i = 0; i < (maxNonZeros); ++i) {
            std::cout << indices[j + m * i] << " ";
        }
        std::cout << std::endl;
    }

/*    std::cout << "indices inline" << std::endl;
    for (int i = 0; i < m * maxNonZeros; ++i) {
        std::cout << i << " = " << indices[i] << std::endl;
    }*/
}
/********************* Main test *********************/

/*
 * Test for non-transposed matrix
 * x is of size n
 * y is of size m
 * a is of size lda * n, where the value of lda must be at least max(1, m).
 */
void testNoTrans() {
    //Actual length of the vectors

    int m = 0, n = 0, maxNonZeros = 0;
    mp_float_t *matrixA;
    int *indices = new int;

    create_ellpack_matrices("/home/ivan/Загрузки/matrixes/5x5 16-not-null.mtx", matrixA, indices, m, n, maxNonZeros);

    int lenx = (1 + (n - 1) * abs(INCX));
    int leny = (1 + (m - 1) * abs(INCY));

    //Inputs
    mp_float_t *vectorX = new mp_float_t[lenx];
    mp_float_t *vectorY = new mp_float_t[leny]();

    for (int i = 0; i < lenx; ++i) {
        mp_set_d(&vectorX[i], (i + 1));
    }

    //Launch tests

    mpres_test(mblas_no_trans, m, n, maxNonZeros, lenx, leny, matrixA, indices, vectorX, vectorY);

    checkDeviceHasErrors(hipDeviceSynchronize());
    // cudaCheckErrors(); //CUMP gives failure

    //Cleanup
    delete[] vectorX;
    delete[] vectorY;
    //delete[] matrixA;

    hipDeviceReset();
}

int main() {

    initialize();

    //Start logging
    Logger::beginTestDescription(Logger::BLAS_SPMV_PERFORMANCE_TEST);
    Logger::printTestParameters(N * M, REPEAT_TEST, MP_PRECISION, MP_PRECISION_DEC);
    Logger::beginSection("Operation info:");
    Logger::printParam("Matrix rows, M", M);
    Logger::printParam("Matrix columns, N", N);
    Logger::printParam("LDA", LDA);
    Logger::printParam("TRANS", TRANS);
    Logger::printDash();
    Logger::beginSection("Additional info:");
    Logger::printParam("RNS_MODULI_SIZE", RNS_MODULI_SIZE);
    Logger::printParam("MPRES_CUDA_BLOCKS_FIELDS_ROUND", MPRES_CUDA_BLOCKS_FIELDS_ROUND);
    Logger::printParam("MPRES_CUDA_THREADS_FIELDS_ROUND", MPRES_CUDA_THREADS_FIELDS_ROUND);
    Logger::printParam("MPRES_CUDA_BLOCKS_RESIDUES", MPRES_CUDA_BLOCKS_RESIDUES);
    Logger::printParam("MPRES_CUDA_THREADS_REDUCE", MPRES_CUDA_THREADS_REDUCE);
    Logger::printParam("CAMPARY_PRECISION (n-double)", CAMPARY_PRECISION);
    Logger::endSection(true);

    //Run the test
    testNoTrans();


    //Finalize
    finalize();

    //End logging
    Logger::endTestDescription();

    return 0;
}