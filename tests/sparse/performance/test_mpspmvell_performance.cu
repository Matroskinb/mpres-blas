#include "hip/hip_runtime.h"
/*
 *  Performance test for SpMV routines using the ELLPACK matrix format
 *
 *  Copyright 2020 by Konstantin Isupov and Ivan Babeshko.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "omp.h"
#include "../../logger.cuh"
#include "../../timers.cuh"
#include "../../tsthelper.cuh"
#include "../../../src/sparse/mpspmvell.cuh"
#include "../../../src/sparse/matrix_converter.cuh"
#include "../../sparse/performance/3rdparty.cuh"

//Execution configuration for mpspmvell
#define MPRES_CUDA_THREADS_SCALAR_KERNELS 64
#define MPRES_CUDA_BLOCKS_RESIDUES 256

#define MATRIX_PATH "../../tests/sparse/matrices/ex40.mtx"
#define MATRIX_SYMMETRIC false

int INP_BITS; //in bits
int INP_DIGITS; //in decimal digits

void setPrecisions() {
    INP_BITS = (int) (MP_PRECISION / 2);
    INP_DIGITS = (int) (INP_BITS / 3.32 + 1);
}

void initialize() {
    hipDeviceReset();
    rns_const_init();
    mp_const_init();
    setPrecisions();
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
}

void finalize() {
}

void convert_vector(double * dest, const mpfr_t *source, int width){
    #pragma omp parallel for
    for( int i = 0; i < width; i++ ){
        dest[i] = mpfr_get_d(source[i], MPFR_RNDN);
    }
}

void convert_vector(mp_float_ptr dest, const mpfr_t *source, int width){
    #pragma omp parallel for
    for( int i = 0; i < width; i++ ){
        mp_set_mpfr(&dest[i], source[i]);
    }
}

void convert_vector(mp_float_ptr dest, const double *source, int width){
    #pragma omp parallel for
    for( int i = 0; i < width; i++ ){
        mp_set_d(&dest[i], source[i]);
    }
}

void print_float(float * data, int size){
    for (int i = 0; i < size; ++i) {
        std::cout << data[i] << " ";
    }
    std::cout<<std::endl;
}

void print_int(int * data, int size){
    for (int i = 0; i < size; ++i) {
        std::cout << data[i] << " ";
    }
    std::cout<<std::endl;
}

/********************* SpMV ELLPACK implementations and benchmarks *********************/

/////////
// double precision
/////////
__global__ static void double_spmv_ell_kernel(const int num_rows, const int cols_per_row, const int * indices, const double * data, const double * x, double * y) {
    unsigned int row = threadIdx.x + blockIdx.x * blockDim.x;
    if(row < num_rows){
        double dot = 0;
        for (int col = 0; col < cols_per_row; col++) {
            int index = indices[col * num_rows + row];
            if(index != -1){
                dot += data[col * num_rows + row] * x[index];
            }
        }
        y[row] += dot;
    }
}

void double_test(const int num_rows, const int num_cols, const int cols_per_row, const double * data, const int * indices, const mpfr_t * x, const mpfr_t * y) {
    InitCudaTimer();
    Logger::printDash();
    PrintTimerName("[GPU] double SpMV ELLPACK");

    //Execution configuration
    int threads = 32;
    int blocks = num_rows / (threads) + (num_rows % (threads) ? 1 : 0);

    //host data
    auto *hx = new double[num_cols];
    auto *hy = new double[num_rows];

    //GPU data
    auto *ddata = new double[num_rows * cols_per_row];
    auto *dindices = new int[num_rows * cols_per_row];
    auto *dx = new double[num_cols];
    auto *dy = new double[num_rows];

    hipMalloc(&ddata, sizeof(double) * num_rows * cols_per_row);
    hipMalloc(&dindices, sizeof(int) * num_rows * cols_per_row);
    hipMalloc(&dx, sizeof(double) * num_cols);
    hipMalloc(&dy, sizeof(double) * num_rows);

    // Convert from MPFR
    convert_vector(hx, x, num_cols);
    convert_vector(hy, y, num_rows);

    //Copying data to the GPU
    hipMemcpy(ddata, data, sizeof(double) * num_rows * cols_per_row, hipMemcpyHostToDevice);
    hipMemcpy(dindices, indices, sizeof(int) * num_rows * cols_per_row, hipMemcpyHostToDevice);
    hipMemcpy(dx, hx, sizeof(double) * num_cols, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, sizeof(double) * num_rows, hipMemcpyHostToDevice);

    //Launch
    StartCudaTimer();
    double_spmv_ell_kernel<<<blocks, threads>>>(num_rows, cols_per_row, dindices, ddata, dx, dy);
    EndCudaTimer();
    PrintCudaTimer("took");
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Copying to the host
    hipMemcpy(hy, dy, sizeof(double) * num_rows , hipMemcpyDeviceToHost);
    print_double_sum(hy, num_rows);

    delete [] hx;
    delete [] hy;
    hipFree(ddata);
    hipFree(dindices);
    hipFree(dx);
    hipFree(dy);
}

/////////
// MPRES-BLAS (structure of arrays)
/////////
void mpres_test(const int num_rows, const int num_cols, const int cols_per_row, const double * data, const int * indices, const mpfr_t * x,  const mpfr_t * y) {
    Logger::printDash();
    InitCudaTimer();
    PrintTimerName("[GPU] MPRES-BLAS mpspmvell");

    size_t matrix_len = num_rows * cols_per_row;
    //Host data
    auto hx = new mp_float_t[num_cols];
    auto hy = new mp_float_t[num_rows];
    auto hdata = new mp_float_t[matrix_len];

    //GPU data
    mp_array_t dx;
    mp_array_t dy;
    mp_array_t ddata;
    mp_array_t dbuf;
    int *dindices;

    //Init data
    cuda::mp_array_init(dx, num_cols);
    cuda::mp_array_init(dy, num_rows);
    cuda::mp_array_init(ddata, matrix_len);
    cuda::mp_array_init(dbuf, matrix_len);
    hipMalloc(&dindices, sizeof(int) * matrix_len);

    // Convert from MPFR and double
    convert_vector(hx, x, num_cols);
    convert_vector(hy, y, num_rows);
    convert_vector(hdata, data, matrix_len);

    //Copying to the GPU
    cuda::mp_array_host2device(dx, hx, num_cols);
    cuda::mp_array_host2device(dy, hy, num_rows);
    cuda::mp_array_host2device(ddata, hdata, matrix_len);
    hipMemcpy(dindices, indices, sizeof(int) * matrix_len, hipMemcpyHostToDevice);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Launch
    StartCudaTimer();
    cuda::mpspmvell<
            MPRES_CUDA_THREADS_SCALAR_KERNELS,
            MPRES_CUDA_BLOCKS_RESIDUES>
            (num_rows, cols_per_row, dindices, ddata, dx, dy, dbuf);
    EndCudaTimer();
    PrintCudaTimer("took");
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Copying to the host
    cuda::mp_array_device2host(hy, dy, num_rows);
    print_mp_sum(hy, num_rows);

    //Cleanup
    delete [] hx;
    delete [] hy;
    delete [] hdata;
    cuda::mp_array_clear(dx);
    cuda::mp_array_clear(dy);
    cuda::mp_array_clear(ddata);
    cuda::mp_array_clear(dbuf);
    hipFree(dindices);
}

/////////
// MPRES-BLAS straightforward (array of structures)
// Each multiple-precision operation is performed by a single thread
/////////
__global__ static void mpspmvell_naive_kernel(const int num_rows, const int cols_per_row, const int * indices, mp_float_ptr data, mp_float_ptr x, mp_float_ptr y) {
    unsigned int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < num_rows) {
        mp_float_t prod;
        mp_float_t dot = cuda::MP_ZERO;
        for (int col = 0; col < cols_per_row; col++) {
            int index = indices[col * num_rows + row];
            if(index >= 0){
                cuda::mp_mul(&prod, &x[index], &data[col * num_rows + row]);
                cuda::mp_add(&dot, &dot, &prod);
            }
        }
        cuda::mp_add(&y[row], &y[row], &dot);
    }
}

void mpres_test_naive(const int num_rows, const int num_cols, const int cols_per_row, const double * data, int * indices, const mpfr_t * x, const mpfr_t * y){
    InitCudaTimer();
    Logger::printDash();
    PrintTimerName("[GPU] MPRES-BLAS mpspmvell (naive)");

    size_t matrix_len = num_rows * cols_per_row;

    //Execution configuration
    int threads = 32;
    int blocks = num_rows / threads + 1;

    // Host data
    auto hx = new mp_float_t[num_cols];
    auto hy = new mp_float_t[num_rows];
    auto hdata = new mp_float_t[matrix_len];

    // GPU data
    mp_float_ptr dx;
    mp_float_ptr dy;
    mp_float_ptr ddata;
    int *dindices;

    //Init data
    hipMalloc(&dx, sizeof(mp_float_t) * num_cols);
    hipMalloc(&dy, sizeof(mp_float_t) * num_rows);
    hipMalloc(&ddata, sizeof(mp_float_t) * matrix_len);
    hipMalloc(&dindices, sizeof(int) * matrix_len);

    // Convert from MPFR
    convert_vector(hx, x, num_cols);
    convert_vector(hy, y, num_rows);
    convert_vector(hdata, data, matrix_len);

    //Copying to the GPU
    hipMemcpy(dx, hx, num_cols * sizeof(mp_float_t), hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, num_rows * sizeof(mp_float_t), hipMemcpyHostToDevice);
    hipMemcpy(ddata, hdata, matrix_len * sizeof(mp_float_t), hipMemcpyHostToDevice);
    hipMemcpy(dindices, indices, matrix_len * sizeof(int), hipMemcpyHostToDevice);

    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Launch
    StartCudaTimer();
    mpspmvell_naive_kernel<<<blocks, threads>>>(num_rows, cols_per_row, dindices, ddata, dx, dy);
    EndCudaTimer();
    PrintCudaTimer("took");
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Copying to the host
    hipMemcpy(hy, dy, num_rows * sizeof(mp_float_t), hipMemcpyDeviceToHost);
    print_mp_sum(hy, num_rows);

    //Cleanup
    delete [] hx;
    delete [] hy;
    delete [] hdata;
    hipFree(dx);
    hipFree(dy);
    hipFree(ddata);
    hipFree(dindices);
}

/////////
// cuSPARSE
/////////
void cusparse_test_coo(const int num_rows, const int num_cols, const int num_lines, mpfr_t *x){
    InitCudaTimer();
    Logger::printDash();
    PrintTimerName("[GPU] cuSPARSE COO");

    int A_num_nnz = num_lines;
    if (IS_SYMMETRIC) {
        A_num_nnz = (num_lines - num_cols) * 2 + num_cols;
    }

/*    int *hA_rows = new int[A_num_nnz];
    int *hA_columns = new int[A_num_nnz];
    float *hA_values = new float[A_num_nnz];*/

    float hA_values[] = { 3.0f, 1.0f, 1.0f, 1.0f,
                          1.0f, 5.0f, 1.0f, 1.0f,
                          1.0f, 1.0f, 7.0f, 1.0f,
                          1.0f, 1.0f, 11.0f, 1.0f,
                          1.0f, 1.0f, 1.0f, 13.0f};
    int hA_rows[] = { 1, 1, 1, 1,
                      2, 2, 2, 2,
                      3, 3, 3, 3,
                      4, 4, 4, 4,
                      5, 5, 5, 5};

    int hA_columns[] ={ 1, 2, 4, 5,
                        1, 2, 3, 5,
                        1, 2, 3, 4,
                        2, 3, 4, 5,
                        1, 3, 4, 5};
    float *hX = new float[num_rows];
    float *hY = new float[num_rows]();


    // Convert from MPFR
    convert_vector(hX, x, num_cols);
    //Convert a sparse matrix to the double-precision ELLPACK format
    convert_to_coo(MATRIX_PATH, num_rows, num_lines, hA_values, hA_rows, hA_columns, IS_SYMMETRIC);

    print_float(hA_values, A_num_nnz);
    print_int(hA_rows, A_num_nnz);
    print_int(hA_columns, A_num_nnz);

    int *dA_rows;
    int *dA_columns;
    float *dA_values;
    float *dX;
    float *dY;
    float alpha = 1.0f;
    float beta = 0.0f;

    hipMalloc(&dA_rows, A_num_nnz * sizeof(int));
    hipMalloc(&dA_columns, A_num_nnz * sizeof(int));
    hipMalloc(&dA_values, A_num_nnz * sizeof(float));
    hipMalloc(&dX, num_cols * sizeof(float));
    hipMalloc(&dY, num_rows * sizeof(float));

    hipMemcpy(dA_rows, hA_rows, A_num_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_columns, hA_columns, A_num_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_values, hA_values, A_num_nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, hX, num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, hY, num_rows * sizeof(float), hipMemcpyHostToDevice);

    hipsparseHandle_t handle = NULL;
    hipsparseStatus_t stat;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX;
    hipsparseDnVecDescr_t vecY;
    void *dBuffer = NULL;
    size_t bufferSize = 0;

    stat = hipsparseCreate(&handle);
    if (stat != HIPSPARSE_STATUS_SUCCESS) {
        printf ("CUSPARSE initialization failed\n");
        return;
    }

    // Create sparse matrix A in COO format
    hipsparseCreateCoo(&matA, num_rows, num_cols, A_num_nnz, dA_rows, dA_columns, dA_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F);
    // Create dense vector X
    hipsparseCreateDnVec(&vecX, num_cols, dX, HIP_R_32F);
    // Create dense vector y
    hipsparseCreateDnVec(&vecY, num_rows, dY, HIP_R_32F);
    // allocate an external buffer if needed
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMV
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    hipMemcpy(hY, dY, num_rows * sizeof(float), hipMemcpyDeviceToHost);
    float sum = 0;
    for (int i = 0; i < num_rows; i++) {
        sum += hY[i];
    }
    printf("result: %.70f\n", sum);

/*    delete [] hA_rows;
    delete [] hA_columns;
    delete [] hA_values;*/
    delete [] hX;
    delete [] hY;
    hipFree(dBuffer);
    hipFree(dA_rows);
    hipFree(dA_columns);
    hipFree(dA_values);
    hipFree(dX);
    hipFree(dY);
}

void cusparse_test_csr(const int num_rows, const int num_cols, const int num_lines, mpfr_t *x) {
    InitCudaTimer();
    Logger::printDash();
    PrintTimerName("[GPU] cuSPARSE CSR");

    int A_num_nnz = num_lines;
    /*int *hA_csrOffsets = new int[num_rows + 1]();
    int *hA_columns = new int[A_num_nnz] ;
    float *hA_values = new float[A_num_nnz];*/

    float hA_values[] = { 3.0f, 1.0f, 1.0f, 1.0f,
                          1.0f, 5.0f, 1.0f, 1.0f,
                          1.0f, 1.0f, 7.0f, 1.0f,
                          1.0f, 1.0f, 11.0f, 1.0f,
                          1.0f, 1.0f, 1.0f, 13.0f};
    int hA_csrOffsets[] = {0, 4, 8, 12, 16, 20};
    int hA_columns[] ={ 1, 2, 4, 5,
                        1, 2, 3, 5,
                        1, 2, 3, 4,
                        2, 3, 4, 5,
                        1, 3, 4, 5};
    float *hX = new float[num_cols];
    float *hY = new float[num_rows]();

    convert_vector(hX, x, num_cols);
    //Convert a sparse matrix to the double-precision ELLPACK format
    //convert_to_csr(MATRIX_PATH, num_rows, num_lines, hA_values, hA_csrOffsets, hA_columns, IS_SYMMETRIC);

    print_float(hA_values, A_num_nnz);
    print_int(hA_csrOffsets, num_rows+1);
    print_int(hA_columns, A_num_nnz);

    int *dA_csrOffsets;
    int *dA_columns;
    float *dA_values;
    float *dX;
    float *dY;
    float alpha = 1.0f;
    float beta = 0.0f;

    hipMalloc((void**) &dA_csrOffsets, (num_rows + 1) * sizeof(int));
    hipMalloc((void**) &dA_columns, A_num_nnz * sizeof(int));
    hipMalloc((void**) &dA_values, A_num_nnz * sizeof(float));
    hipMalloc((void**) &dX, num_cols * sizeof(float));
    hipMalloc((void**) &dY, num_rows * sizeof(float));

    hipMemcpy(dA_csrOffsets, hA_csrOffsets, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_columns, hA_columns, A_num_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dA_values, hA_values, A_num_nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, hX, num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dY, hY, num_rows * sizeof(float), hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t handle = NULL;
    hipsparseStatus_t stat;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX;
    hipsparseDnVecDescr_t vecY;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    hipsparseCreate(&handle);
    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, num_rows, num_cols, A_num_nnz, dA_csrOffsets, dA_columns, dA_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F);
    // Create dense vector X
    hipsparseCreateDnVec(&vecX, num_cols, dX, HIP_R_32F);
    // Create dense vector y
    hipsparseCreateDnVec(&vecY, num_rows, dY, HIP_R_32F);
    // allocate an external buffer if needed
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute SpMV
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

    // destroy matrix/vector descriptors
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    //--------------------------------------------------------------------------
    // device result check

    hipMemcpy(hY, dY, num_rows * sizeof(float), hipMemcpyDeviceToHost);
    float sum = 0;
    for (int i = 0; i < num_rows; i++) {
        sum += hY[i];
    }
    printf("result: %.70f\n", sum);

    //--------------------------------------------------------------------------
    // device memory deallocation
/*    delete [] hA_csrOffsets;
    delete [] hA_columns;
    delete [] hA_values;*/
    delete [] hX;
    delete [] hY;
    hipFree(dBuffer);
    hipFree(dA_csrOffsets);
    hipFree(dA_columns);
    hipFree(dA_values);
    hipFree(dX);
    hipFree(dY);
}

/********************* Main test *********************/

void test( int NUM_ROWS, int NUM_COLS, int NUM_LINES, int COLS_PER_ROW) {

    //Inputs
    mpfr_t *vectorX = create_random_array(NUM_COLS, INP_BITS);
    mpfr_t *vectorY = create_random_array(NUM_ROWS, INP_BITS);
    auto * data = new double [NUM_ROWS * COLS_PER_ROW]();
    auto * indices = new int[NUM_ROWS * COLS_PER_ROW]();

    //Convert a sparse matrix to the double-precision ELLPACK format
    convert_to_ellpack(MATRIX_PATH, NUM_ROWS, COLS_PER_ROW, NUM_LINES, data, indices, MATRIX_SYMMETRIC);

    //TODO: Delete after debugging
 /*
    for (int i = 0; i < NUM_COLS; ++i) {
        mpfr_set_si(vectorX[i], (i+1), MPFR_RNDN);
    }

    print_ellpack(NUM_ROWS,COLS_PER_ROW,data,indices);
    int count = 0;
    for (int i = 0; i < NUM_ROWS * COLS_PER_ROW; ++i) {
        if (data[i] != 0) {
            count++;
        }
    }
    std::cout<<std::endl<<"NonZeros: "<<count<<std::endl;
*/
    //Launch tests
    double_test(NUM_ROWS, NUM_COLS, COLS_PER_ROW, data, indices, vectorX, vectorY);
    mpres_test(NUM_ROWS, NUM_COLS, COLS_PER_ROW, data, indices, vectorX, vectorY);
    mpres_test_naive(NUM_ROWS, NUM_COLS, COLS_PER_ROW, data, indices, vectorX, vectorY);
    campary_spmv_ell_test<CAMPARY_PRECISION>(NUM_ROWS, NUM_COLS, COLS_PER_ROW, data, indices, vectorX, vectorY, INP_DIGITS);
    cump_spmv_ell_test(NUM_ROWS, NUM_COLS, COLS_PER_ROW, data, indices, vectorX, vectorY, MP_PRECISION, INP_DIGITS);

    checkDeviceHasErrors(hipDeviceSynchronize());
    // cudaCheckErrors(); //CUMP gives failure

    //Cleanup
    for(int i = 0; i < NUM_COLS; i++){
        mpfr_clear(vectorX[i]);
    }
    for(int i = 0; i < NUM_ROWS; i++){
        mpfr_clear(vectorY[i]);
    }
    delete[] vectorX;
    delete[] vectorY;
    delete[] data;
    delete[] indices;
    hipDeviceReset();
}

int main() {
    //The operation parameters. Read from an input file that contains a sparse matrix
    int NUM_ROWS = 0; //number of rows
    int NUM_COLS = 0; //number of columns
    int NUM_LINES = 0; //number of lines in the input matrix file
    int COLS_PER_ROW = 0; //maximum number of nonzeros per row

    initialize();

    //Start logging
    Logger::beginTestDescription(Logger::BLAS_SPMV_ELL_PERFORMANCE_TEST);
    Logger::beginSection("Operation info:");
    Logger::printParam("Matrix path", MATRIX_PATH);
    read_matrix_properties(MATRIX_PATH, NUM_ROWS, NUM_COLS, NUM_LINES, COLS_PER_ROW, MATRIX_SYMMETRIC);
    Logger::printParam("Matrix rows, NUM_ROWS", NUM_ROWS);
    Logger::printParam("Matrix columns, NUM_COLUMNS", NUM_COLS);
    Logger::printParam("Maximum nonzeros per row, COLS_PER_ROW", COLS_PER_ROW);
    Logger::printDash();
    Logger::beginSection("Additional info:");
    Logger::printParam("RNS_MODULI_SIZE", RNS_MODULI_SIZE);
    Logger::printParam("MPRES_CUDA_BLOCKS_FIELDS_ROUND", MPRES_CUDA_THREADS_SCALAR_KERNELS);
    Logger::printParam("MPRES_CUDA_THREADS_FIELDS_ROUND", MPRES_CUDA_BLOCKS_RESIDUES);
    Logger::printParam("CAMPARY_PRECISION (n-double)", CAMPARY_PRECISION);
    Logger::endSection(true);

    //Run the test
    test(NUM_ROWS, NUM_COLS, NUM_LINES, COLS_PER_ROW);

    //Finalize
    finalize();

    //End logging
    Logger::endTestDescription();

    return 0;
}